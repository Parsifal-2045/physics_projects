#include "hip/hip_runtime.h"
// C++ standard headers
#include <cassert>
#include <iostream>
#include <vector>

// CUDA headers
#include <hip/hip_runtime.h>

// local headers
#include "cuda_check.h"

// Here you can set the device ID that was assigned to you
#define MYDEVICE 0

// Part 3 of 5: implement the kernel
__global__ void myFirstKernel(int *a)
{
  auto index = threadIdx.x + blockIdx.x * blockDim.x;
  a[index] = index + 42;
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
  CUDA_CHECK(hipSetDevice(MYDEVICE));

  // Create a CUDA stream to execute asynchronous operations on this device
  hipStream_t queue;
  CUDA_CHECK(hipStreamCreate(&queue));

  // Your problem's size
  int N = 64;

  // Define the grid and block size
  int numThreadsPerBlock = 8;
  int numBlocks = (N + numThreadsPerBlock - 1) / numThreadsPerBlock;

  // Allocate and initialize host memory
  // hint: the vector is empty, you might want to allocate some memory ...
  std::vector<int> h_a;
  h_a.reserve(N);

  // Pointer for the device memory
  int *d_a;

  // Part 1 of 5: allocate the device memory
  size_t memSize = N * sizeof(int);
  CUDA_CHECK(hipMallocAsync(&d_a, memSize, queue));

  // Part 2 of 5: configure and launch kernel
  myFirstKernel<<<numBlocks, numThreadsPerBlock, 0, queue>>>(d_a);
  // Check for any errors that occurred during kernel launch
  CUDA_CHECK(hipGetLastError());

  // Part 4 of 5: copy data from device to host asynchronously
  CUDA_CHECK(hipMemcpyAsync(h_a.data(), d_a, memSize, hipMemcpyDeviceToHost, queue));

  // Free the device memory
  CUDA_CHECK(hipFreeAsync(d_a, queue));

  // Wait for all asynchronous operations to complete
  CUDA_CHECK(hipStreamSynchronize(queue));

  // Part 5 of 5: verify that the data returned to the host is correct
  for (int i = 0; i < N; ++i)
  {
    assert(h_a[i] == i + 42);
  }

  // Destroy the CUDA stream
  CUDA_CHECK(hipStreamDestroy(queue));

  // If the program makes it this far, then the results are correct and
  // there are no run-time errors.  Good work!
  std::cout << "Correct, good work!" << std::endl;
}
