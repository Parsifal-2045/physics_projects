
#include <hip/hip_runtime.h>
#include <iostream>
#include <cassert>

__global__ void add(const int *a, const int *b, int *c, const int n)
{
    auto index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n)
    {
        c[index] = a[index] + b[index];
    }
}

int main()
{
    long long int N = (1 << 28) + 1;
    hipStream_t stream;
    hipStreamCreate(&stream);
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    size_t size = N * sizeof(int);

    // Allocate pinned memory on host
    hipHostMalloc(&a, size, hipHostMallocDefault);
    hipHostMalloc(&b, size, hipHostMallocDefault);
    hipHostMalloc(&c, size, hipHostMallocDefault);
    for (int i = 0; i != N; ++i)
    {
        a[i] = i;
        b[i] = 8 * i;
    }

    // Allocate memory on device
    hipMallocAsync(&d_a, size, stream);
    hipMallocAsync(&d_b, size, stream);
    hipMallocAsync(&d_c, size, stream);

    // Copy inputs to device
    hipMemcpyAsync(d_a, a, size, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_b, b, size, hipMemcpyHostToDevice, stream);

    // Launch kernel on GPU
    // Tell the scheduler what resources each block needs
    // <<<nBlocks, nThreads per block, max amount of shared memory to use, CUDA stream>>>
    int nThreadsPerBlock = 512;
    int nBlocks = (N + nThreadsPerBlock - 1) / nThreadsPerBlock;
    size_t maxDinamicSharedMem = 0;
    add<<<nBlocks, nThreadsPerBlock, maxDinamicSharedMem, stream>>>(d_a, d_b, d_c, N);

    // Retrieve results
    hipMemcpyAsync(c, d_c, size, hipMemcpyDeviceToHost, stream);

    // Free device memory
    hipFreeAsync(d_a, stream);
    hipFreeAsync(d_b, stream);
    hipFreeAsync(d_c, stream);

    // Synch to be able to use c on host
    hipStreamSynchronize(stream);
    for (int i = 0; i != N; ++i)
    {
        assert(a[i] + b[i] == c[i]);
        // std::cout << a[i] << "+" << b[i] << "=" << c[i] << '\n';
    }
    std::cout << "Correct result!" << '\n';

    // Destroy the stream and free pinned host memory
    hipStreamDestroy(stream);
    hipHostFree(a);
    hipHostFree(b);
    hipHostFree(c);
}