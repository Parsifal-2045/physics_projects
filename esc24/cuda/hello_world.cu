
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void mykernel()
{
    printf("Hello world!\n");
}

int main()
{
    hipStream_t stream;
    hipStreamCreate(&stream);
    mykernel<<<1, 1, 0>>>();
    hipStreamSynchronize(stream);
    hipStreamDestroy(stream);
}