#include "hip/hip_runtime.h"
// C++ standard headers
#include <cassert>
#include <iostream>
#include <vector>

// CUDA headers
#include <hip/hip_runtime.h>

// local headers
#include "cuda_check.h"

// Here you can set the device ID that was assigned to you
#define MYDEVICE 0

// Part 2 of 4: implement the kernel
__global__ void kernel(int *a, int dimx, int dimy)
{
  auto row = threadIdx.y + blockIdx.y * blockDim.y;
  auto col = threadIdx.x + blockIdx.x * blockDim.x;
  auto linear_index = row * dimx + col;
  if (linear_index < dimx * dimy)
  {
    a[linear_index] = linear_index;
  }
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main()
{
  CUDA_CHECK(hipSetDevice(MYDEVICE));

  // Create a CUDA stream to execute asynchronous operations on this device
  hipStream_t queue;
  CUDA_CHECK(hipStreamCreate(&queue));

  // Part 1 and 4 of 4: set the dimensions of the matrix
  int dimx = 19;
  int dimy = 67;

  // Allocate enough memory on the host
  std::vector<int> h_a(dimx * dimy);
  int memSize = dimx * dimy * sizeof(int);

  // Pointer for the device memory
  int *d_a;

  // Allocate enough memory on the device
  CUDA_CHECK(hipMallocAsync(&d_a, memSize, queue));

  // Part 2 of 4: define grid and block size and launch the kernel
  dim3 numberOfBlocks, numberOfThreadsPerBlock;
  numberOfThreadsPerBlock.x = std::min(32, dimx);
  numberOfThreadsPerBlock.y = std::min(32, dimy);
  numberOfBlocks.x = (dimx + numberOfThreadsPerBlock.x - 1) / numberOfThreadsPerBlock.x;
  numberOfBlocks.y = (dimy + numberOfThreadsPerBlock.y - 1) / numberOfThreadsPerBlock.y;

  kernel<<<numberOfBlocks, numberOfThreadsPerBlock, 0, queue>>>(d_a, dimx, dimy);
  CUDA_CHECK(hipGetLastError());

  // Device to host copy
  CUDA_CHECK(hipMemcpyAsync(h_a.data(), d_a, memSize, hipMemcpyDeviceToHost, queue));

  // Free the device memory
  CUDA_CHECK(hipFreeAsync(d_a, queue));

  // Wait for all asynchronous operations to complete
  CUDA_CHECK(hipStreamSynchronize(queue));

  // verify the data returned to the host is correct
  for (int row = 0; row < dimy; ++row)
  {
    for (int col = 0; col < dimx; ++col)
      assert(h_a[row * dimx + col] == row * dimx + col);
  }

  // Destroy the CUDA stream
  CUDA_CHECK(hipStreamDestroy(queue));

  // If the program makes it this far, then the results are correct and
  // there are no run-time errors.  Good work!
  std::cout << "Correct!" << std::endl;

  return 0;
}
